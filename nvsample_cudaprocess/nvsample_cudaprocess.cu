#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "customer_functions.h"
#include "cudaEGL.h"
#include "iva_metadata.h"

#define BOX_W 32
#define BOX_H 32

#define CORD_X 64
#define CORD_Y 64
#define MAX_BUFFERS 30
static BBOX rect_data[MAX_BUFFERS];

/**
  * Dummy custom pre-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
  /* add your custom pre-process here
     we draw a green block for demo */
  int x, y;
  char * uv = NULL;
  unsigned char * rgba = NULL;
  if (sformat[1] == COLOR_FORMAT_U8_V8) {
    uv = (char *)sBaseAddr[1];
    for (y = 0; y < BOX_H; ++y) {
      for (x = 0; x < BOX_W; ++x) {
        uv[y * spitch[1] + 2 * x] = 0;
        uv[y * spitch[1] + 2 * x + 1] = 0;
      }
    }
  } else if (sformat[0] == COLOR_FORMAT_RGBA) {
    rgba = (unsigned char *)sBaseAddr[0];
     for (y = 0; y < BOX_H*2; y++) {
      for (x = 0; x < BOX_W*8; x+=4) {
       rgba[x + 0] = 0;
       rgba[x + 1] = 0;
       rgba[x + 2] = 0;
       rgba[x + 3] = 0;
      }
        rgba+=spitch[0];
    }
  }
}

/**
  * Dummy custom post-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
  /* add your custom post-process here
     we draw a green block for demo */
  int x, y;
  char * uv = NULL;
  int xoffset = (CORD_X * 4);
  int yoffset = (CORD_Y * 2);
  unsigned char * rgba = NULL;
  if (sformat[1] == COLOR_FORMAT_U8_V8) {
    uv = (char *)sBaseAddr[1];
    for (y = 0; y < BOX_H; ++y) {
      for (x = 0; x < BOX_W; ++x) {
        uv[(y + BOX_H * 2) * spitch[1] + 2 * (x + BOX_W * 2)] = 0;
        uv[(y + BOX_H * 2) * spitch[1] + 2 * (x + BOX_W * 2) + 1] = 0;
      }
    }
  } else if (sformat[0] == COLOR_FORMAT_RGBA) {
    rgba = (unsigned char *)sBaseAddr[0];
    rgba += ((spitch[0] * yoffset) + xoffset);
     for (y = 0; y < BOX_H*2; y++) {
      for (x = 0; x < BOX_W*8; x+=4) {
       rgba[(x + xoffset) + 0] = 0;
       rgba[(x + xoffset) + 1] = 0;
       rgba[(x + xoffset) + 2] = 0;
       rgba[(x + xoffset) + 3] = 0;
      }
        rgba+=spitch[0];
    }
  }
}

__global__ void addLabelsKernel(int* pDevPtr, int pitch){
  int row = blockIdx.y*blockDim.y + threadIdx.y + BOX_H;
  int col = blockIdx.x*blockDim.x + threadIdx.x + BOX_W;
  char * pElement = (char*)pDevPtr + row * pitch + col * 2;
  pElement[0] = 0;
  pElement[1] = 0;
  return;
}

static int addLabels(hipDeviceptr_t pDevPtr, int pitch){
    dim3 threadsPerBlock(BOX_W, BOX_H);
    dim3 blocks(1,1);
    addLabelsKernel<<<blocks,threadsPerBlock>>>((int*)pDevPtr, pitch);
    return 0;
}

static void add_metadata(void ** usrptr)
{
    /* User need to fill rectangle data based on their requirement.
     * Here rectangle data is filled for demonstration purpose only */

    int i;
    static int index = 0;

    rect_data[index].framecnt = index;
    rect_data[index].objectcnt = index;

    for(i=0; i < NUM_LOCATIONS; i++)
    {
        rect_data[index].location_list[i].x1 = index;
        rect_data[index].location_list[i].x2 = index;
        rect_data[index].location_list[i].y1 = index;
        rect_data[index].location_list[i].y2 = index;
    }
    *usrptr = &rect_data[index];
    index++;
    if(!(index % MAX_BUFFERS))
    {
        index = 0;
    }
}

/**
  * Performs CUDA Operations on egl image.
  *
  * @param image : EGL image
  */
static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }

  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_ABGR) {
    /* Rectangle label in plane RGBA, you can replace this with any cuda algorithms */
      addLabels((hipDeviceptr_t) eglFrame.frame.pPitch[0], eglFrame.pitch);
    } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
    /* Rectangle label in plan UV , you can replace this with any cuda algorithms */
      addLabels((hipDeviceptr_t) eglFrame.frame.pPitch[1], eglFrame.pitch);
    } else
      printf ("Invalid eglcolorformat\n");
  }

  add_metadata(usrptr);

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }
}

extern "C" void
init (CustomerFunction * pFuncs)
{
  pFuncs->fPreProcess = pre_process;
  pFuncs->fGPUProcess = gpu_process;
  pFuncs->fPostProcess = post_process;
}

extern "C" void
deinit (void)
{
  /* deinitialization */
}
